#include "hip/hip_runtime.h"
#include "OpenGLWnd.h"
#include <hip/hip_runtime_api.h>
#include <drvapi_error_string.h>
#include <cuda_gl_interop.h>




std::vector<CudaDevice> CDataBridge::DevicesList;

inline bool ErrorControl(hipError_t Status)
{
	if (Status != hipSuccess)
	{
		qDebug("cuda error");
		return false;
	}
	return true;
}


__global__ void FillVBOTest(GLubyte *Buffer, VertexData *Buffer2);

bool compareByVersion(const CudaDevice &a, const CudaDevice &b)
{
	return a.nDeviceVersion < b.nDeviceVersion;
}

int CDataBridge::InitCudaDevice(OpenGLWnd* wnd){
	int			nDeviceMajor = 0;
	int			nDeviceMinor = 0;
	int			nDeviceCounter = 0;

	hipError_t ErrorId = hipInit(0);
	if (ErrorId != hipSuccess) {
		QMessageBox::information(
			wnd,
			"Init",
			"Can't Init CUDA Framework");
		return 0;
	}

	ErrorId = hipGetDeviceCount(&nDeviceCounter);
	if (ErrorId != hipSuccess) {
		QString str = "hipGetDeviceCount returns an error: " + QString(getCudaDrvErrorString(ErrorId));

		QMessageBox::information(
			wnd,
			"Init",
			str);
		return 0;
	}
	if (nDeviceCounter == 0) {
		QMessageBox::information(
			wnd,
			"Init",
			"Can't find CUDA supported Device");
		return 0;
	}

	for (hipDevice_t CudaDev = 0; CudaDev < nDeviceCounter; CudaDev++)
	{
		ErrorId = hipDeviceComputeCapability(&nDeviceMajor, &nDeviceMinor, CudaDev);
		if (ErrorId != hipSuccess) {
			QString str = "hipDeviceComputeCapability returns an error: " + QString(getCudaDrvErrorString(ErrorId));
			QMessageBox::information(
				wnd,
				"Init",
				str);
			return 0;
		}
		if (nDeviceMajor = MinimumRequireMajor && nDeviceMinor >= MinimumRequireMinor || nDeviceMajor > MinimumRequireMajor)
			DevicesList.push_back(CudaDevice(CudaDev, nDeviceMajor * 100 + nDeviceMinor));
	}
	if (DevicesList.size() == 0){
		QMessageBox::information(
			wnd,
			"Init",
			"Cuda version on device is too low ");
		return 0;
	}

	std::sort(DevicesList.begin(), DevicesList.end(), compareByVersion);

	return 1;
}

void CDataBridge::ConnectToDevice(){
	qDebug("connect to cuda dev");
	CudaDevIt nCurrentDevice;

	if (DevicesList.size() == 0){
		qDebug("Init Devices List");
		if (!InitCudaDevice(parent)) return exit(1);

		qDebug("Init Devices List Done");
	}

	nCurrentDevice = DevicesList.begin();

	hipSetDevice(nCurrentDevice->DeviceId);
	//cudaGLSetGLDevice(nCurrentDevice->DeviceId);
	char strDeviceName[256];
	hipError_t ErrorId = hipDeviceGetName(strDeviceName, 256, nCurrentDevice->DeviceId);
	if (ErrorId != hipSuccess) {
		qDebug("Cuda Devices Get Name Error Done");
		QString str = "hipDeviceGetName returns an error: " + QString(getCudaDrvErrorString(ErrorId));
		QMessageBox::information(
			parent,
			"Cuda Init",
			str);
		exit(1);
	}
	QString str = QString(strDeviceName) + " is used by application";
	emit showStatusText(str);

}
void CDataBridge::PrepareVBO(){
	//qDebug("return from PrepareVBO"); return;

	const int width = 2048;
	const int height = 1024;

	int nGridSize = width * height;
	
	

	// wygenerowanie dw�ch identyfikator�w obiekt�w buforowych
	//if (vbo_id[0] != -1){
	//	parent->glDeleteBuffers(3, vbo_id);
	//	parent->glDeleteVertexArrays(1, my_vao);
	//}
	parent->glGenVertexArrays(1, my_vao);
	parent->glBindVertexArray(my_vao[0]);
	parent->glGenBuffers(2, vbo_id);

	parent->glBindBuffer(GL_ARRAY_BUFFER, vbo_id[0]);
	parent->glBufferData(GL_ARRAY_BUFFER, nGridSize * 12, NULL, GL_DYNAMIC_DRAW);				/////zmiana

	size_t* size = NULL;
	parent->glEnableVertexAttribArray(0);
	parent->glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, (GLvoid*)0);

	ErrorControl(hipGraphicsGLRegisterBuffer(resource1, vbo_id[0], hipGraphicsRegisterFlagsWriteDiscard));
	ErrorControl(hipGraphicsMapResources(1, resource1));
	ErrorControl(hipGraphicsResourceGetMappedPointer((void**)&vertexData, size, resource1[0]));
	ErrorControl(hipGraphicsUnmapResources(1, resource1));

	parent->glBindBuffer(GL_ARRAY_BUFFER, vbo_id[1]);
	parent->glBufferData(GL_ARRAY_BUFFER, nGridSize * 4, NULL, GL_DYNAMIC_DRAW);				/////zmiana



	parent->glEnableVertexAttribArray(1);
	parent->glVertexAttribPointer(1, 4, GL_UNSIGNED_BYTE, GL_TRUE, 0, (GLvoid*)0);

	ErrorControl(hipGraphicsGLRegisterBuffer(resource2, vbo_id[1], hipGraphicsRegisterFlagsWriteDiscard));
	ErrorControl(hipGraphicsMapResources(1, resource2));
	ErrorControl(hipGraphicsResourceGetMappedPointer((void**)&vertexColorData, size, resource2[0]));
	ErrorControl(hipGraphicsUnmapResources(1, resource2));
	//	GLvoid * buf = glMapBufferRange(GL_ARRAY_BUFFER, 0, GridSize * 3, GL_MAP_WRITE_BIT);

	// skopiowanie danych do drugiego obiektu buforowego
	//	memcpy(buf, vertcol, GridSize * 3);

	// zwolnienie wska�nika na dane drugiego obiektu buforowego
	//	if (glUnmapBuffer(GL_ARRAY_BUFFER) == GL_FALSE)
	//	{
	//		printf("Niepoprawne odwzorowanie obiektu buforowego\n");
	//		exit(0);
	//	}
	ErrorControl(hipGraphicsMapResources(1, resource2));
	ErrorControl(hipGraphicsMapResources(1, resource1));
	FillVBOTest << <width, height, 0 >> >(vertexColorData, vertexData);
	ErrorControl(hipGraphicsUnmapResources(1, resource2));
	ErrorControl(hipGraphicsUnmapResources(1, resource1));
	//FillVBOBuffer();

	
	//ExitIfCudaGLError("ConstructVAO");

	
	// FillVBOBuffer();
	//*/
	IndexesNumber = 6 * (width - 1)*(height - 1);
	GLuint* Indexes = new unsigned int[IndexesNumber];

	for (int i = 0; i < (width - 1); i++){
		for (int j = 0; j < (height - 1); j++){
			Indexes[6 * (i*(height - 1) + j)] = j + height * i;
			Indexes[6 * (i*(height - 1) + j) + 1] = j + height * (i + 1);
			Indexes[6 * (i*(height - 1) + j) + 2] = j + 1 + height * (i + 1);
			Indexes[6 * (i*(height - 1) + j) + 3] = j + 1 + height * i;
			Indexes[6 * (i*(height - 1) + j) + 4] = j + height * i;
			Indexes[6 * (i*(height - 1) + j) + 5] = j + 1 + height * (i + 1);
		}
	}
	parent->glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vbo_id[2]);
	parent->glBufferData(GL_ELEMENT_ARRAY_BUFFER, IndexesNumber * sizeof(GLuint), Indexes, GL_STATIC_DRAW);


	delete[] Indexes;
}
void CDataBridge::paintVAO(){
	parent->glBindVertexArray(my_vao[0]);
	parent->glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vbo_id[2]);


	parent->glDrawElements(
		GL_TRIANGLES,      // mode
		IndexesNumber,    // count

		GL_UNSIGNED_INT,   // type
		(void*)0           // element array buffer offset
		);


	parent->glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
}
__global__ void FillVBOTest(GLubyte *Buffer, VertexData *Buffer2){

	int y = threadIdx.x;
	int x = blockIdx.x;
	
	int id = y + x*1024;

	float posx = (float)(x - 1024.0f) / 1024.0f;
	float posy = (float)(y - 512.0f) / 512.0f;

	Buffer2[id].x = posx;
	Buffer2[id].y = posy;
	Buffer2[id].z = -sqrt(posx*posx + posy*posy);

	Buffer[4 * (id)] =  GLubyte((1 - sqrt(posx*posx + posy*posy)) * 255);
	Buffer[4 * (id)+1] = 0;
	Buffer[4 * (id)+2] = 0;
	Buffer[4 * (id)+3] = 255;
}
